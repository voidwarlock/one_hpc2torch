#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
const int TM = 8;
const int TN = 8;
const int BLOCK_DIM_x = 16;
const int BLOCK_DIM_y = 16;
const int BM = TM * BLOCK_DIM_x;
const int BN = TN * BLOCK_DIM_y;
const int BK = 8;
//
#include <mma.h>
using namespace nvcuda;
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 8;
const int warpSize = 32;
const int warpNum = BLOCK_DIM_x * BLOCK_DIM_y / warpSize;
const int warpX = (warpNum == 1 ? 1 : 2);
const int warpY = warpNum / warpX;
template <int BM, int BN, int BK, int TM, int TN>
__global__ void matrixKernel5th(float *dA, float *dB, float *dC, int M, int K, int N)
{
    __shared__ float SA[BM * BK * 2];
    __shared__ float SB[BK * BN * 2];
    int indA = TM * (blockIdx.x * blockDim.x);
    int indB = TN * (blockIdx.y * blockDim.y);
    int width = (K + BK - 1) / BK;
    float tmp[TM * TN] = {0.0f};
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int smem_a_m = tid / 2;
    int smem_a_k = tid % 2;
    int smem_b_k = tid / 32;
    int smem_b_n = tid % 32;
    float a[4];
    float b[4];
    float com_a[TM];
    float com_b[TN];
    //------------
    int ph = 0;
    (float4 &)a[0] = (float4 &)dA[(indA + smem_a_m) * K + 4 * smem_a_k + ph * BK];
    SA[(4 * smem_a_k) * BM + smem_a_m + ph % 2 * BM * BK] = a[0];
    SA[(4 * smem_a_k + 1) * BM + smem_a_m + ph % 2 * BM * BK] = a[1];
    SA[(4 * smem_a_k + 2) * BM + smem_a_m + ph % 2 * BM * BK] = a[2];
    SA[(4 * smem_a_k + 3) * BM + smem_a_m + ph % 2 * BM * BK] = a[3];
    (float4 &)b[0] = (float4 &)dB[(smem_b_k + ph * BK) * N + indB + 4 * smem_b_n];
    (float4 &)SB[smem_b_k * BN + 4 * smem_b_n] = (float4 &)b[0];

    __syncthreads();

    for (int ph = 1; ph < width; ph++)
    {
        (float4 &)a[0] = (float4 &)dA[(indA + smem_a_m) * K + 4 * smem_a_k + ph * BK];
        (float4 &)b[0] = (float4 &)dB[(smem_b_k + ph * BK) * N + indB + 4 * smem_b_n];

        //-------------
        for (int index_k = 0; index_k < BK; index_k++)
        {
            (float4 &)com_a[0] = (float4 &)SA[index_k * BM + threadIdx.x * TM + (ph - 1) % 2 * BM * BK];
            (float4 &)com_a[4] = (float4 &)SA[index_k * BM + threadIdx.x * TM + 4 + (ph - 1) % 2 * BM * BK];
            (float4 &)com_b[0] = (float4 &)SB[index_k * BN + threadIdx.y * TN + (ph - 1) % 2 * BN * BK];
            (float4 &)com_b[4] = (float4 &)SB[index_k * BN + threadIdx.y * TN + 4 + (ph - 1) % 2 * BN * BK];
            for (int index_q = 0; index_q < TM; index_q++)
            {
                for (int index_v = 0; index_v < TN; index_v++)
                {
                    tmp[index_q * TN + index_v] += com_a[index_q] * com_b[index_v];
                }
            }
        }
        SA[(4 * smem_a_k) * BM + smem_a_m + ph % 2 * BM * BK] = a[0];
        SA[(4 * smem_a_k + 1) * BM + smem_a_m + ph % 2 * BM * BK] = a[1];
        SA[(4 * smem_a_k + 2) * BM + smem_a_m + ph % 2 * BM * BK] = a[2];
        SA[(4 * smem_a_k + 3) * BM + smem_a_m + ph % 2 * BM * BK] = a[3];

        (float4 &)SB[smem_b_k * BN + 4 * smem_b_n + ph % 2 * BN * BK] = (float4 &)b[0];
        __syncthreads();
    }
    //--------------
    ph = width;
    for (int index_k = 0; index_k < BK; index_k++)
    {
        (float4 &)com_a[0] = (float4 &)SA[index_k * BM + threadIdx.x * TM + (ph - 1) % 2 * BM * BK];
        (float4 &)com_a[4] = (float4 &)SA[index_k * BM + threadIdx.x * TM + 4 + (ph - 1) % 2 * BM * BK];
        (float4 &)com_b[0] = (float4 &)SB[index_k * BN + threadIdx.y * TN + (ph - 1) % 2 * BN * BK];
        (float4 &)com_b[4] = (float4 &)SB[index_k * BN + threadIdx.y * TN + 4 + (ph - 1) % 2 * BN * BK];
        for (int index_q = 0; index_q < TM; index_q++)
        {
            for (int index_v = 0; index_v < TN; index_v++)
            {
                tmp[index_q * TN + index_v] += com_a[index_q] * com_b[index_v];
            }
        }
    }
    for (int index_q = 0; index_q < TM; index_q++)
    {
        for (int index_v = 0; index_v < TN; index_v++)
        {
            int reg_c_m = threadIdx.x * TM + index_q;
            int reg_c_n = threadIdx.y * TN + index_v;
            if (indA + index_q < M && indB + index_v < N)
            {
                dC[(indA + reg_c_m) * N + indB + reg_c_n] = tmp[index_q * TN + index_v];
            }
        }
    }
}
__global__ void row_wmma_ker(float *dA, float *dB, float *dC, int M, int K, int N)
{
    int lda = K; // A=[M,K],索引(x,y) = x * K + y，列优先原则索引(x,y) = y * M + x
    int ldb = N;
    int ldc = N;

    int indA = blockIdx.x * warpX * WMMA_M;
    int indB = blockIdx.y * warpY * WMMA_N;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int warpId = tid / warpSize;
    int warpIdx = warpId % warpX;
    int warpIdy = warpId / warpX;

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> left_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> right_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    // Initialize the output to zero
    wmma::fill_fragment(c_frag, 0.0f);
    int aRow = indA + warpIdx * WMMA_M;
    int bCol = indB + warpIdy * WMMA_N;
    int width = (K + WMMA_K - 1) / WMMA_K;
    for (int i = 0; i < width; i++)
    {
        int aCol = i * WMMA_K;
        int bRow = i * WMMA_K;
        if (aRow < M && aCol < K && bRow < K && bCol < N)
        {
            // 读取A,B矩阵里面子矩阵的元素
            wmma::load_matrix_sync(left_frag, dA + aRow * lda + aCol, lda);
            wmma::load_matrix_sync(right_frag, dB + bRow * ldb + bCol, ldb);
            // 子矩阵做乘法
            wmma::mma_sync(c_frag, left_frag, right_frag, c_frag);
        }
    }
    int cRow = aRow;
    int cCol = bCol;
    if (cRow < M && cCol < N)
    {
        // Store the output
        wmma::store_matrix_sync(dC + cRow * ldc + cCol, c_frag, ldc, wmma::mem_row_major);
    }
}
extern "C" void matmul_cuda_f32(void const *dA, void const *dB, void *dC, int M, int K, int N)
{

    int num_blocks_x = (M + BM - 1) / BM;
    int num_blocks_y = (N + BN - 1) / BN;
    dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
    dim3 grid_dim(num_blocks_x, num_blocks_y, 1);
    matrixKernel5th<BM, BN, BK, TM, TN><<<grid_dim, block_dim>>>((float *)dA, (float *)dB, (float *)dC, M, K, N);

    // int num_block_x = (M + WMMA_M * warpX - 1) / (WMMA_M * warpX);
    // int num_block_y = (N + WMMA_N * warpY - 1) / (WMMA_N * warpY);

    // dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
    // dim3 grid_dim(num_block_x, num_block_y, 1);
    // row_wmma_ker<<<grid_dim, block_dim>>>((float *)dA, (float *)dB, (float *)dC, M, K, N);
}
